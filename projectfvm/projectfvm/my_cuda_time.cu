
#include <hip/hip_runtime.h>
#include <stdio.h>

struct MyCudaTime {

	hipEvent_t _start;
	hipEvent_t _beforeKernel, _afterKernel, _stop;

	MyCudaTime() {
		hipEventCreate(&_start);
		hipEventCreate(&_beforeKernel);
		hipEventCreate(&_afterKernel);
		hipEventCreate(&_stop);

		hipEventRecord(_start, 0);
	}

	void beforeKernel() {
		hipEventRecord(_beforeKernel, 0);
	}
	
	void afterKernel() {
		hipEventRecord(_afterKernel, 0);
	}

	void stop() {  // return elapsed time in milliseconds
		hipEventRecord(_stop, 0);
		hipEventSynchronize(_stop);
	}

	void report() {
		float elapsedTime;

		hipEventElapsedTime(&elapsedTime, _start, _stop);
		printf("Total time %3.2f ms\n", elapsedTime);   // why 3.1?

		hipEventElapsedTime(&elapsedTime, _start, _beforeKernel);
		printf("\t Before calling kernel %3.2f ms\n", elapsedTime); 

		hipEventElapsedTime(&elapsedTime, _beforeKernel, _afterKernel);
		printf("\t In kernel %3.2f ms\n", elapsedTime); 

		hipEventElapsedTime(&elapsedTime, _afterKernel, _stop);
		printf("\t After calling kernel %3.2f ms\n", elapsedTime); 
	}
};